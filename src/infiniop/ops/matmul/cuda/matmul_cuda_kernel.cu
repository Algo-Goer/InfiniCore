#include "../../utils.h"
#include "./matmul_cuda.cuh"

template<typename Tdata>
infiniopStatus_t cudaMatmulCublas(infiniopMatmulCudaDescriptor_t desc, void *c, float beta, void const *a, void const *b, float alpha, void *stream) {
    auto info = desc->info;

    if (info.is_transed) {
        std::swap(a, b);
    }

    hipDataType a_type, b_type, c_type;
    hipblasComputeType_t compute_type;
    if constexpr (std::is_same<Tdata, half>::value) {
        a_type = b_type = c_type = HIP_R_16F;
        compute_type = HIPBLAS_COMPUTE_32F;
    } else {
        a_type = b_type = c_type = HIP_R_32F;
#ifdef ENABLE_SUGON_CUDA_API
        compute_type = HIPBLAS_COMPUTE_32F;
#else
        compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
#endif
    }

    auto op_a = info.a_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    auto op_b = info.b_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    use_cublas(desc->cublas_handles_t, desc->device_id, (hipStream_t) stream,
               [&](hipblasHandle_t handle) { hipblasGemmStridedBatchedEx(
                                                handle,
                                                op_a,
                                                op_b,
                                                static_cast<int>(info.m),
                                                static_cast<int>(info.n),
                                                static_cast<int>(info.k),
                                                &alpha,
                                                a,
                                                a_type,
                                                static_cast<int>(info.a_matrix.ld()),
                                                info.a_matrix.stride,
                                                b,
                                                b_type,
                                                static_cast<int>(info.b_matrix.ld()),
                                                info.b_matrix.stride,
                                                &beta,
                                                c,
                                                c_type,
                                                static_cast<int>(info.c_matrix.ld()),
                                                info.c_matrix.stride,
                                                static_cast<int>(info.batch),
                                                compute_type,
                                                CUBLAS_GEMM_DEFAULT_TENSOR_OP); });
    return INFINIOP_STATUS_SUCCESS;
}

infiniopStatus_t cudaMatmul(infiniopMatmulCudaDescriptor_t desc,
                            void *workspace,
                            uint64_t workspace_size,
                            void *c,
                            void const *a,
                            void const *b,
                            float alpha,
                            float beta,
                            void *stream) {
    if (desc->dtype == INFINI_DTYPE_F16) {
        return cudaMatmulCublas<half>(desc, c, beta, a, b, alpha, stream);
    }
    if (desc->dtype == INFINI_DTYPE_F32) {
        return cudaMatmulCublas<float>(desc, c, beta, a, b, alpha, stream);
    }
    return INFINIOP_STATUS_BAD_TENSOR_DTYPE;
}
