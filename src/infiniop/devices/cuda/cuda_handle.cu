#include "common_cuda.cuh"

infiniStatus_t createCudaHandle(infiniopCudaHandle_t *handle_ptr, infiniDevice_t cuda_device_type) {
    // Create a new cublas handle pool
    int device_id = 0;
    CHECK_CUDA_OR_RETURN(hipGetDevice(&device_id), INFINI_STATUS_DEVICE_NOT_INITIALIZED);
    auto pool = std::make_shared<Pool<hipblasHandle_t>>();
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    pool->push(std::move(handle));

    // create a cudnn handle pool
    auto cudnn_pool = std::make_shared<Pool<hipdnnHandle_t>>();
    hipdnnHandle_t cudnn_handle;
    CHECK_CUDNN(hipdnnCreate(&cudnn_handle));
    cudnn_pool->push(std::move(cudnn_handle));

    // set CUDA device property
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

    // set device compute capability numbers
    int capability_major;
    int capability_minor;
    hipDeviceGetAttribute(&capability_major, hipDeviceAttributeComputeCapabilityMajor, device_id);
    hipDeviceGetAttribute(&capability_minor, hipDeviceAttributeComputeCapabilityMinor, device_id);

    *handle_ptr = new InfiniopCudaHandle{
        cuda_device_type,
        device_id,
        std::move(pool),
        std::move(cudnn_pool),
        std::move(prop),
        capability_major,
        capability_minor,
    };

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t destroyCudaHandle(infiniopCudaHandle_t handle_ptr) {
    handle_ptr->cublas_handle_pool = nullptr;
    handle_ptr->cudnn_handle_pool = nullptr;
    delete handle_ptr;

    return INFINI_STATUS_SUCCESS;
}
