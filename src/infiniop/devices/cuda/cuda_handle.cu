#include "./common_cuda.cuh"

infiniopStatus_t createCudaHandle(infiniopCudaHandle_t *handle_ptr, int device_id, infiniDevice_t cuda_device_type) {
    // Check if device_id is valid
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_id >= device_count) {
        return INFINIOP_STATUS_BAD_DEVICE;
    }

    // Create a new cublas handle pool
    auto pool = std::make_shared<Pool<hipblasHandle_t>>();
    if (hipSetDevice(device_id) != hipSuccess) {
        return INFINIOP_STATUS_BAD_DEVICE;
    }
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    pool->push(std::move(handle));

    // create a cudnn handle pool
    auto cudnn_pool = std::make_shared<Pool<hipdnnHandle_t>>();
    hipdnnHandle_t cudnn_handle;
    checkCudnnError(hipdnnCreate(&cudnn_handle));
    cudnn_pool->push(std::move(cudnn_handle));

    // set CUDA device property
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

    // set device compute capability numbers
    int capability_major;
    int capability_minor;
    hipDeviceGetAttribute(&capability_major, hipDeviceAttributeComputeCapabilityMajor, device_id);
    hipDeviceGetAttribute(&capability_minor, hipDeviceAttributeComputeCapabilityMinor, device_id);

    *handle_ptr = new InfiniopCudaHandle{
        cuda_device_type,
        device_id,
        std::move(pool),
        std::move(cudnn_pool),
        std::move(prop),
        capability_major,
        capability_minor,
    };

    return INFINIOP_STATUS_SUCCESS;
}

infiniopStatus_t destroyCudaHandle(infiniopCudaHandle_t handle_ptr) {
    handle_ptr->cublas_handles_t = nullptr;
    handle_ptr->cudnn_handles_t = nullptr;
    delete handle_ptr;

    return INFINIOP_STATUS_SUCCESS;
}
